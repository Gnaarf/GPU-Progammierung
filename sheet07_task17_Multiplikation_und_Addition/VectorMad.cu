#include "hip/hip_runtime.h"

// Includes
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

// Variables
float* h_A;
float* h_B;
float* h_C;
float* h_D;
float* h_E;
float* d_A;
float* d_B;
float* d_C;
float* d_D;
float* d_E;

// Functions
void Cleanup(void);
void RandomInit(float*, int);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, float* D, float* E)
{
    int i = threadIdx.x;    
    C[i] = A[i] + B[i];	
	E[i] = A[i] + B[i] * D[i];
}

// Host code
int main(int argc, char** argv)
{
    printf("Simple vector addition\n");
    int N = 256;
    size_t size = N * sizeof(float);    

    // Allocate input vectors h_A, h_B and h_C in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) Cleanup();
    h_B = (float*)malloc(size);
    if (h_B == 0) Cleanup();
    h_C = (float*)malloc(size);
    if (h_C == 0) Cleanup();
	h_D = (float*)malloc(size);
    if (h_D == 0) Cleanup();
	h_E = (float*)malloc(size);
    if (h_E == 0) Cleanup();
	
    // Initialize input vectors
    RandomInit(h_A, N);
    RandomInit(h_B, N);	
	RandomInit(h_D, N);
	
    // Allocate vectors in device memory
    CUDA_SAFE_CALL( hipMalloc((void**)&d_A, size) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_B, size) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_C, size) );	
	CUDA_SAFE_CALL( hipMalloc((void**)&d_D, size) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_E, size) );

    // Copy vectors from host memory to device memory
    CUDA_SAFE_CALL( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(d_D, h_D, size, hipMemcpyHostToDevice) );

	// Invoke kernel
	VecAdd<<<1, N>>>(d_A, d_B, d_C, d_D, d_E);
	
#ifdef _DEBUG
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
#endif


	// Copy result from device memory to host memory
    // h_C contains the result in host memory
    CUDA_SAFE_CALL( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(h_E, d_E, size, hipMemcpyDeviceToHost) );

    // Verify result
	// TODO: Print out E and verify the result.
    int i = 0;
    for (i = 0; i < N; ++i) 
	{
        float sum = h_A[i] + h_B[i];
		float product = h_A[i] + h_B[i] * h_D[i];
		printf("%f + %f * %f = %f\n", h_A[i], h_B[i], h_D[i], h_E[i]);
        if (fabs(h_E[i] - product) > 1e-5)
            break;
    }
    printf("%s \n", (i == N) ? "PASSED" : "FAILED");
    
    Cleanup();
}

void Cleanup(void)
{
    // Free device memory
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

	// TODO: Free device memory of D and E.
	if (d_D)
        hipFree(d_D);
    if (d_E)
        hipFree(d_E);


    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);

	// TODO: Free host memory of D and E.	
	if (h_D)
        free(h_D);
    if (h_E)
        free(h_E);

    CUDA_SAFE_CALL( hipDeviceReset() );
        
    printf("\nPress ENTER to exit...\n");
    fflush( stdout);
    fflush( stderr);
    getchar();    

    exit(0);
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}
