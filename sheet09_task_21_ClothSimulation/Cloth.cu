#include "hip/hip_runtime.h"

// Includes
#include "CudaMath.h"
#include "Cloth.h"

// Computes the impacts between two points that are connected by a constraint in order to satisfy the constraint a little better.
__device__ float3 computeImpact(float3 me, float3 other, float stepsize, float h)
{	
	const float aimedDistance = 1.0 / (float)RESOLUTION_X;
	float3 dir = other-me;
	float ldir = length(dir);
	if (ldir==0) return dir;
	float e = (ldir - aimedDistance) * 0.5;
	float3 debug = dir/ldir * e / (h*h) * stepsize; 
	return debug;
}

// Simple collision detection against a sphere at (0,0,0) with radius SPHERE_RADIUS and skin width SKIN_WIDTH.
__device__ float3 sphereCollision(float3 p, float h)
{
	// TODO: Testen, ob Punkt im inneren der Kugel ist. Wenn ja, dann einen Impuls berechnen, der sie wieder heraus bewegt.
	float centerDist = length(p);
	if(centerDist < SPHERE_RADIUS+SKIN_WIDTH)
	{
		return (p)/h;
	}
	else
	{
		return make_float3(0,0,0);
	}
	//return p;
}

// -----------------------------------------------------------------------------------------------
// Aufsummieren der Impulse, die von den benachbarten Gitterpunkten ausge�bt werden.
// impacts += ...
__global__ void computeImpacts(float3* oldPos, float3* impacts, float stepsize, float h)
{
	// TODO: Positionen der benachbarten Gitterpunkte und des eigenen Gitterpunktes ablesen.
	int pointX = blockIdx.x * blockDim.x + threadIdx.x;
	int pointY = blockIdx.y * blockDim.y + threadIdx.y;
	int myIndex = pointX * RESOLUTION_Y + pointY;
	int up = pointY -1;
	int down = pointY +1;
	int left = pointX -1;
	int right = pointX +1;

	float3 impactsTmp = make_float3(0, 0, 0);

	// TODO: Kollisionsbehandlung mit Kugel durchf�hren.
	impacts[myIndex] += sphereCollision(oldPos[myIndex],h);

	// TODO: Mit jedem Nachbar besteht ein Constraint. Dementsprechend f�r jeden Nachbar 
	//		 computeImpact aufrufen und die Ergebnisse aufsummieren.
	if(up > 0)
	{
		int upIdx = pointX * RESOLUTION_Y + up;
		float3 tmp = computeImpact(oldPos[myIndex], oldPos[upIdx],stepsize,h);
		impactsTmp = impactsTmp + tmp;
	}
	if(down < RESOLUTION_Y)
	{
		int downIdx = pointX * RESOLUTION_Y + down;
		float3 tmp = computeImpact(oldPos[myIndex], oldPos[downIdx],stepsize,h);
		impactsTmp = impactsTmp + tmp;
	}
	if(left > 0)
	{
		int leftIdx = left * RESOLUTION_Y + pointY;
		float3 tmp = computeImpact(oldPos[myIndex], oldPos[leftIdx],stepsize,h);
		impactsTmp = impactsTmp + tmp; 
	}
	if(right < RESOLUTION_X)
	{
		int rightIdx = right * RESOLUTION_Y + pointY;
		float3 tmp = computeImpact(oldPos[myIndex], oldPos[rightIdx],stepsize,h);
		impactsTmp = impactsTmp + tmp;
	}

	// TODO: Die Summe der Impulse auf "impacts" des eigenen Gitterpunkts addieren.	
	bool debug = !(impactsTmp.x == 0 && impactsTmp.y == 0 && impactsTmp.z == 0);
	if(debug)
	{
		impacts[myIndex] = impacts[myIndex] + impactsTmp;
	}
}

// -----------------------------------------------------------------------------------------------
// Preview-Step
// newpos = oldpos + (velocity + impacts * h) * h
__global__ void previewSteps(	float3* newPos, float3* oldPos, float3* impacts, float3* velocity,								
								float h)
{
	int pointX = blockIdx.x * blockDim.x + threadIdx.x;
	int pointY = blockIdx.y * blockDim.y + threadIdx.y;
	int index = pointX * RESOLUTION_Y + pointY;
	// TODO: Berechnen, wo wir w�ren, wenn wir eine Integration von der bisherigen Position 
	//		 mit der bisherigen Geschwindigkeit und den neuen Impulsen durchf�hren.
	newPos[index] = oldPos[index] + velocity[index] * h + impacts[index] * h*h;
	
	//newPos[index] = oldPos[index] - make_float3(0,0.002,0);
}

// -----------------------------------------------------------------------------------------------
// Integrate velocity
// velocity = velocity * LINEAR_DAMPING + (impacts - (0,GRAVITY,0)) * h 
__global__ void integrateVelocity(	float3* impacts, float3* velocity, float h)
{
	int pointX = blockIdx.x * blockDim.x + threadIdx.x;
	int pointY = blockIdx.y * blockDim.y + threadIdx.y;
	int index = pointX * RESOLUTION_Y + pointY;
	// TODO: Update velocity.
	float3 debug = velocity[index] * LINEAR_DAMPING + (impacts[index] - make_float3(0,GRAVITY,0)) * h; 
	velocity[index] = debug;
}

// -----------------------------------------------------------------------------------------------
// Test-Funktion die nur dazu da ist, damit man etwas sieht, sobald die VBOs gemapped werden...
__global__ void test( float3* newPos, float3* oldPos, float h)
{
	newPos[blockIdx.x] = oldPos[blockIdx.x] + make_float3(0, -h, 0);
}

__global__ void computeNormals(float3* pos, float3* normals)
{
	//int myIndex = blockIdx.x * RESOLUTION_Y + blockIdx.y;
	int pointX = blockIdx.x * blockDim.x + threadIdx.x;
	int pointY = blockIdx.y * blockDim.y + threadIdx.y;
	int myIndex = pointX * RESOLUTION_Y + pointY;
	int up = int(pointY) -1;
	int down = int(pointY) +1;
	int left = int(pointX) -1;
	int right = int(pointX) +1;

	float3 upv = make_float3(0,0,0);
	float3 downv = make_float3(0,0,0);
	float3 leftv = make_float3(0,0,0);
	float3 rightv = make_float3(0,0,0);

	if(up < 0){
		int downIdx = pointX*RESOLUTION_Y+down;
		upv = -1 * pos[downIdx]-pos[myIndex];
	}
	else{
		int upIdx = pointX*RESOLUTION_Y+up;
		upv = pos[upIdx]-pos[myIndex];
	}

	if(down >= RESOLUTION_Y){
		int upIdx = pointX*RESOLUTION_Y+up;
		downv = -1 * pos[upIdx]-pos[myIndex];
	}
	else{
		int downIdx = pointX*RESOLUTION_Y+down;
		downv = pos[down]-pos[myIndex];
	}

	if(left < 0){
		int rightIdx = right*RESOLUTION_Y+pointY;
		leftv = -1 * pos[rightIdx]-pos[myIndex];
	}
	else{
		int leftIdx = left*RESOLUTION_Y+pointY;
		leftv = pos[leftIdx]-pos[myIndex];
	}

	if(right >= RESOLUTION_X)
	{
		int leftIdx = left*RESOLUTION_Y+pointY;
		rightv = -1 * pos[leftIdx] - pos[myIndex];
	}
	else
	{
		int rightIdx = right*RESOLUTION_Y+pointY;
		rightv = pos[rightIdx]-pos[myIndex];
	}

	float3 n1 = cross(rightv, upv);
	float3 n2 = cross(leftv, downv);

	/*float3 dsad = make_float3(0,1,0);

	if(pointX == 63 && pointY == 63)
	{
		normals[myIndex] = dsad;
	}
	else
	{
		normals[myIndex] = make_float3(1,0,0);
	}*/

	normals[myIndex] = -1*(n1+n2)/2;

}

// -----------------------------------------------------------------------------------------------
void updateCloth(	float3* newPos, float3* oldPos, float3* impacts, float3* velocity, float3* normals,					
					float h, float stepsize)
{
	// dont move the row resY
	dim3 gridSize(RESOLUTION_X,RESOLUTION_Y-1);
	dim3 blockSIze(1, 1);

	dim3 normalGridSize(RESOLUTION_X,RESOLUTION_Y);

	// -----------------------------
	// Clear impacts
	hipMemset(impacts, 0, RESOLUTION_X*RESOLUTION_Y*sizeof(float3));

	// Updating constraints is an iterative process.
	// The more iterations we apply, the stiffer the cloth become.
	for (int i=0; i<10; ++i)
	{
		// -----------------------------
		// TODO: previewSteps Kernel aufrufen (Vorhersagen, wo die Gitterpunkte mit den aktuellen Impulsen landen w�rden.)
		// newpos = oldpos + (velocity + impacts * h) * h		
		previewSteps<<<gridSize,blockSIze>>>(newPos, oldPos,impacts,velocity,h);
		// -----------------------------
		// TODO: computeImpacts Kernel aufrufen (Die Impulse neu berechnen, sodass die Constraints besser eingehalten werden.)
		// impacts += ...
		computeImpacts<<<gridSize,blockSIze>>>(newPos,impacts,stepsize,h);
	}

	// -----------------------------
	// TODO: Approximieren der Normalen
	computeNormals<<<normalGridSize,blockSIze>>>(newPos, normals);

	// -----------------------------
	// TODO: Integrate velocity kernel ausf�hren
	// Der kernel berechnet:  velocity = velocity * LINEAR_DAMPING + (impacts - (0,GRAVITY,0)) * h 	
	integrateVelocity<<<gridSize,blockSIze>>>(impacts,velocity,h);
	//previewSteps<<<blocks,1>>>(newPos, oldPos,impacts,velocity,h);
}